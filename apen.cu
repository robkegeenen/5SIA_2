#include "hip/hip_runtime.h"
#include "eeg.h"

__device__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__
void apen_correlation (int np, int32_t *x, unsigned int m, double r, double *result)
{
  bool set;
  unsigned int count;
  __shared__ double sum;
  sum = 0;
  unsigned i = threadIdx.x;
  //printf("BlockID: %d\n", i);
  //for(unsigned int i = 0; i <= np - m; i++){
    count = 0;
    __syncthreads();
    for(unsigned int j = 0; j <= np - m; j++){
      set = false;
      for(unsigned int k = 0; k < m; k++){
        if(abs(x[i + k] - x[j + k]) > r){
          set = true;
          break;
        }
      }
      if(!set){
        count++;
      }
    }
    atomicAdd(&sum, ((double)count) / ((double)np - m + 1));
    //sum += ((double)count) / ((double)np - m + 1);
    //if(threadIdx.x == 0){
      printf("Sum: %lf\n", sum);
      //}
    //}
  __syncthreads();
  if(i == 0){
  *result = sum / ((double)np - m + 1);
  }
}

void apen(int np, int32_t *x, float *a, unsigned int m, double r)
{
  double *dev_inter1, inter1, *dev_inter2, inter2;
  int32_t *dev_x;
  cudaCheckError(hipMalloc(&dev_x, np*sizeof(int32_t)));
  cudaCheckError(hipMalloc(&dev_inter1, sizeof(double)));
  cudaCheckError(hipMalloc(&dev_inter2, sizeof(double)));
  cudaCheckError(hipMemcpy(dev_x, x, np*sizeof(int32_t), hipMemcpyHostToDevice));
  apen_correlation<<<1, (np - m + 1)>>>(np, dev_x, m, r, dev_inter1);
  apen_correlation<<<1, (np - m + 1)>>>(np, dev_x, m + 1, r, dev_inter2);
  cudaCheckError(hipDeviceSynchronize());
  cudaCheckError(hipMemcpy(&inter1, dev_inter1, sizeof(double), hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(&inter2, dev_inter2, sizeof(double), hipMemcpyDeviceToHost));
  cudaCheckError(hipFree(dev_x));
  cudaCheckError(hipFree(dev_inter1));
  cudaCheckError(hipFree(dev_inter2));
  *a = log(inter1 / inter2);
}
