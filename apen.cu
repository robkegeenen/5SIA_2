#include "hip/hip_runtime.h"
#include "eeg.h"

__device__ double atomicAdd(double* address, double val){
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do{
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  }while(assumed != old);
  return __longlong_as_double(old);
}

__global__
void apen_correlation (int np, int32_t *x, unsigned int m, double r, double *result)
{
  unsigned int i = blockIdx.x;
  unsigned int j = threadIdx.x;
  bool set;
  __shared__ unsigned int count;
  if(i == 0){
    *result = 0;
  }
  count = 0;
  __syncthreads();
  set = false;
  for(unsigned int k = 0; k < m; k++){
    if(abs(x[i + k] - x[j + k]) > r){
      set = true;
      break;
    }
  }
  if(!set){
    atomicAdd(&count, 1);
  }
  __syncthreads();
  if(j == 0){
    atomicAdd(result, ((double)count) / ((double)np - m + 1));
  }
}

void apen(int np, int32_t *x, float *a, unsigned int m, double r)
{
  double *dev_inter1, inter1, *dev_inter2, inter2;
  int32_t *dev_x;
  int length1 = np - (m + 0) + 1;
  int length2 = np - (m + 1) + 1;
  hipStream_t stream1, stream2; //Only helps a little bit
  cudaCheckError(hipStreamCreate(&stream1));
  cudaCheckError(hipStreamCreate(&stream2));
  cudaCheckError(hipMalloc(&dev_x, np*sizeof(int32_t)));
  cudaCheckError(hipMalloc(&dev_inter1, sizeof(double)));
  cudaCheckError(hipMalloc(&dev_inter2, sizeof(double)));
  cudaCheckError(hipMemcpy(dev_x, x, np*sizeof(int32_t), hipMemcpyHostToDevice));
  apen_correlation<<<length1, length1, 0, stream1>>>(np, dev_x, m + 0, r, dev_inter1);
  apen_correlation<<<length2, length2, 0, stream2>>>(np, dev_x, m + 1, r, dev_inter2);
  cudaCheckError(hipDeviceSynchronize());
  cudaCheckError(hipMemcpy(&inter1, dev_inter1, sizeof(double), hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(&inter2, dev_inter2, sizeof(double), hipMemcpyDeviceToHost));
  cudaCheckError(hipFree(dev_x));
  cudaCheckError(hipFree(dev_inter1));
  cudaCheckError(hipFree(dev_inter2));
  *a = log((inter1 / ((double)length1)) / (inter2 / ((double)length2)));
}
